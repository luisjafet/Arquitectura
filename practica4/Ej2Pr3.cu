#include "hip/hip_runtime.h"
/*
 *  Ejercicio 2 Práctica 3: CUDA
 *  Desempeño en función de la homogeneidad para acceder a memoria
 *  y de la regularidad del código
 */

#include <stdio.h>
//PP#include <hip/hip_runtime.h>

#define STRIDE       8
#define OFFSET        1
#define GROUP_SIZE  8

/* Utilidad para checar errores de CUDA */
void checkCUDAError(const char*);

// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N, int *ept, int *bsi, int *tsi, int *tei, int *gr)
{
	int n_elem_per_thread = N / (gridDim.x * blockDim.x);
	int block_start_idx = n_elem_per_thread * blockIdx.x * blockDim.x;
	int thread_start_idx = block_start_idx
	+ (threadIdx.x / STRIDE) * n_elem_per_thread * STRIDE
	+ ((threadIdx.x + OFFSET) % STRIDE);
	int thread_end_idx = thread_start_idx + n_elem_per_thread * STRIDE;
	if(thread_end_idx > N) thread_end_idx = N;
	int group = (threadIdx.x / GROUP_SIZE) & 1;
	*ept=n_elem_per_thread;
	*bsi=block_start_idx;
	*tsi=thread_start_idx;
	*tei=thread_end_idx;
	*gr=group;
	for(int idx=thread_start_idx; idx < thread_end_idx; idx+=STRIDE)
	{
		if(!group) a[idx] = a[idx] * a[idx];
		else       a[idx] = a[idx] + a[idx];
	}
}

// main routine that executes on the host
int main(void)
{
	float *a_h, *a_d;  // Pointer to host & device arrays
	const int N = 1<<10;  // Make a big array with 2**N elements
	size_t size = N * sizeof(float);
    int  *ept,  *bsi,  *tsi,  *tei,  *gr;
    /* Auxiliares para medir tiempos */
    hipEvent_t start, stop;
    float time;

    a_h = (float *)malloc(size);        // Allocate array on host
	hipMalloc((void **) &a_d, size);   // Allocate array on device
	hipMalloc((void**) &ept, sizeof(int));
	hipMalloc((void**) &bsi, sizeof(int));
	hipMalloc((void**) &tsi, sizeof(int));
	hipMalloc((void**) &tei, sizeof(int));
	hipMalloc((void**) &gr, sizeof(int));
	
    // Initialize host array and copy it to CUDA device
	for (int i=0; i<N; i++)
        a_h[i] = (float)i;

	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    checkCUDAError("memcpy");

	// Create timer for timing CUDA calculation
	//PPunsigned int timer = 0;
	//PPcutCreateTimer( &timer );
	hipEventCreate(&start);
	hipEventCreate(&stop);
    
    // Set number of threads and blocks
	int n_threads_per_block = 128;//1<<9;  // 512 threads per block
	int n_blocks = 256;//1<<10;  // 1024 blocks

	// Do calculation on device

    hipEventRecord(start,0);
	square_array <<< n_blocks, n_threads_per_block >>> (a_d, N, ept, bsi, tsi, tei,gr);
	hipDeviceSynchronize();  // Wait for square_array to finish on CUDA

    checkCUDAError("kernel invocation");


	// Retrieve result from device and store it in host array
	hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);
    checkCUDAError("memcpy");

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime( &time, start, stop );

	// Print some of the results
	for (int i=0; i<N; i+=N/50) printf("%d %f\n", i, a_h[i]);

    // Imprime tiempo de ejecución
    printf("\n\nTIEMPO DE EJECUCIÓN: %f mSeg\n\n", time);
	printf("Valor de N: %d\n", N);
	int group = (128 / GROUP_SIZE) & 1;
	printf("\ngroup: %d\n", group);
	
	hipMemcpy(&group, ept, sizeof(int), hipMemcpyDeviceToHost);
	printf("\nept: %d", group);
	hipMemcpy(&group, bsi, sizeof(int), hipMemcpyDeviceToHost);
	printf("\nbsi: %d", group);
	hipMemcpy(&group, tsi, sizeof(int), hipMemcpyDeviceToHost);
	printf("\ntsi: %d", group);
	hipMemcpy(&group, tei, sizeof(int), hipMemcpyDeviceToHost);
	printf("\ntei: %d", group);
	hipMemcpy(&group, gr, sizeof(int), hipMemcpyDeviceToHost);
	printf("\ngroup: %d\n", group);

    hipEventDestroy( start );
    hipEventDestroy( stop );

	free(a_h); hipFree(a_d);
}

/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}